#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include "hipfft/hipfft.h"
#include <hipsparse.h>
#include "hip/hip_complex.h"
#include<hipblas.h>
#include<hipsparse.h>
#define  NX 500

__device__  bool flag = true;
//hipError_t TestFFT(hipComplex* idata, hipComplex* odata);
hipError_t TestFFT(hipComplex* idata,hipComplex* odata)
{
	/*int* cdata_dev = 0;
	if (hipMalloc((void**)&cdata_dev, sizeof(int) * 1024 * 1024 * 1024) != hipSuccess)
		printf("CUDA MALLOC CDATA FAILED!\n");
	hipFree(cdata_dev);*/
	hipfftHandle plan;
	hipfftComplex *data;


	hipMalloc((void**)&data, sizeof(hipfftComplex)*NX);
	hipMemcpy(data, idata, sizeof(hipfftComplex)*NX,hipMemcpyHostToDevice);
	
	
	if (hipfftPlan1d(&plan,NX,HIPFFT_C2C,1)!=HIPFFT_SUCCESS)
	{
		std::cout << "Plan creation failed" << std::endl;
	}
	
	

	if (hipfftExecC2C(plan,data,data,HIPFFT_FORWARD)!=HIPFFT_SUCCESS)
	{
		std::cout << "ExecC2C Forward failed" << std::endl;
	}
	hipDeviceSynchronize();
	
	/*if (hipDeviceSynchronize()!=hipSuccess)
	{
		std::cout << "Failed to synchronized" << std::endl;
	}*/
	hipMemcpy(odata, data, sizeof(hipfftComplex)*NX, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);

	return hipSuccess;
}




//int TestCuSparse()
//{
//	hipComplex Znear[8];
//	int Z_near_row[5];
//	int Z_near_col[8];
//	int Z_near_length = 8;
//	Z_near_row[0] = 0;
//	Z_near_row[1] = 2;
//	Z_near_row[2] = 4;
//	Z_near_row[3] = 6;
//	Z_near_row[4] = 8;
//
//	Z_near_col[0] = 0;
//	Z_near_col[1] = 2;
//	Z_near_col[2] = 0;
//	Z_near_col[3] = 1;
//	Z_near_col[4] = 1;
//	Z_near_col[5] = 3;
//	Z_near_col[6] = 1;
//	Z_near_col[7] = 3;
//	Znear[0].x = 1.0f; Znear[0].y = 1;
//	Znear[1].x = 1.0f; Znear[1].y = 3.0;
//	Znear[2].x = 1.0f; Znear[2].y = -1.0f;
//	Znear[3].x = 2.0f; Znear[3].y = 1.0f;
//	Znear[4].x = 3.0f; Znear[4].y = 4.0f;
//	Znear[5].x = 2.0f; Znear[5].y = 1.0f;
//	Znear[6].x = 1.0f; Znear[6].y = 5.0f;
//	Znear[7].x = 4.0f; Znear[7].y = 5.0f;
//
//	hipComplex V[4];
//	V[0].x = 1; V[0].y = 1;
//	V[1].x = 2; V[1].y = 3;
//	V[2].x = 3; V[2].y = 4;
//	V[3].x = 4; V[3].y = 5;
//
//	hipComplex V1[4];
//	V1[0].x = 2; V1[0].y = 3;
//	V1[1].x = 1; V1[1].y = 7;
//	V1[2].x = 3; V1[2].y = 4;
//	V1[3].x = 5; V1[3].y = 9;
//
//
//
//
//	hipComplex alpha = { 1.0f, 0.0f };
//	hipComplex beta = { 0.0f, 0.0f };
//	hipsparseHandle_t handle;
//	hipsparseMatDescr_t descr;
//	hipsparseCreate(&handle);
//	hipsparseCreateMatDescr(&descr);
//
//	hipComplex* Znear_dev = 0;
//	int* Znear_row_dev = 0;
//	int* Znear_col_dev = 0;
//	hipComplex* V_dev = 0;
//	hipComplex* res_dev = 0;
//	hipMalloc((void**)&Znear_dev, sizeof(hipComplex) * 8);
//	hipMalloc((void**)&Znear_row_dev, sizeof(int) * 5);
//	hipMalloc((void**)&Znear_col_dev, sizeof(int) * 8);
//	hipMalloc((void**)&V_dev, sizeof(hipComplex) * 4);
//	hipMalloc((void**)&res_dev, sizeof(hipComplex) * 4);
//	hipMemcpy(Znear_dev, Znear, sizeof(hipComplex) * 8,hipMemcpyHostToDevice);
//	hipMemcpy(Znear_row_dev, Z_near_row, sizeof(int) * 5, hipMemcpyHostToDevice);
//	hipMemcpy(Znear_col_dev, Z_near_col, sizeof(int) * 8, hipMemcpyHostToDevice);
//	hipMemcpy(V_dev, V, sizeof(hipComplex) * 4, hipMemcpyHostToDevice);
//	hipsparseCcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 8, &alpha, descr, Znear_dev, Znear_row_dev, Znear_col_dev, V_dev, &beta, res_dev);
//	hipMemcpy(V, res_dev, sizeof(hipComplex) * 4, hipMemcpyDeviceToHost);
//
//	for (int i = 0; i < 4;i++)
//	{
//		printf("%lf+%lfi\n", V[i].x, V[i].y);
//	}
//
//	hipMemcpy(V_dev, V1, sizeof(hipComplex) * 4, hipMemcpyHostToDevice);
//	hipsparseCcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 8, &alpha, descr, Znear_dev, Znear_row_dev, Znear_col_dev, V_dev, &beta, res_dev);
//	hipMemcpy(V1, res_dev, sizeof(hipComplex) * 4, hipMemcpyDeviceToHost);
//
//	for (int i = 0; i < 4; i++)
//	{
//		printf("%lf+%lfi\n", V1[i].x, V1[i].y);
//	} 
//
//
//
//
//
//
//
//
//
//
//	hipsparseDestroy(handle);
//	hipsparseDestroyMatDescr(descr);
//
//
//	return 0;
//}


void TestcublasCdotc()
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipComplex x[3] = { { 1, 2 }, { 2, 3 }, { 3, 4 } };
	hipComplex y[3] = { { 3, 1 }, { 1, 4 }, { 1, 2 } };
	hipComplex tmp[3];
	//for (int i = 0; i < 4;i++)
	//{
	//	printf("%f,%f\n", x[i].x, x[i].y);
	//}
	hipComplex *x_dev = 0;
	hipComplex *y_dev = 0;
	if (hipMalloc((void**)&x_dev, sizeof(hipComplex) * 3) != hipSuccess ||
		hipMalloc((void**)&y_dev, sizeof(hipComplex) * 3) != hipSuccess)
		printf("cuda malloc failed!\n");
	if (hipMemcpy(x_dev, x, sizeof(hipComplex) * 3, hipMemcpyHostToDevice) != hipSuccess ||
		hipMemcpy(y_dev, y, sizeof(hipComplex) * 3, hipMemcpyHostToDevice) != hipSuccess)
		printf("cuda memcpy failed!\n");
	/*hipblasSetVector(3, sizeof(hipComplex), x, 1, x_dev, 1);
	hipblasSetVector(3, sizeof(hipComplex), y, 1, y_dev, 1);*/
	hipDeviceSynchronize();
	
	/*for (int i = 0; i < 4; i++){
		printf("%f,%f\n", tmp[i].x, tmp[i].y);
	}*/
	//hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	hipComplex result;

	hipblasCdotc(handle, 3, x_dev, 1, y_dev, 1, &result);
	
	hipDeviceSynchronize();
	//hipMemcpy(result, result_dev, sizeof(hipComplex), hipMemcpyDeviceToHost);
	printf("%20.15f,%20.15f\n", result.x,result.y);
	hipblasDestroy(handle);
}




__global__ void f1_kernel(int* arr, int size){
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	//if (id == 0)
		//printf("call f1_kernel\n");
	if (id < size){
		//while (flag){}
		int k = 1;
		int j = 100000000;
		int res = 9;
		while (k){
			while (j>0){
				j--;
				res = res*k;
			}
			k--;
			//printf("k:%d\n", k);
		}
		k = INT_MAX;
		j = INT_MAX;
		/*while (k + j){
			k--;
			j--;
			res = res*k + 1 * res*res + res - res * 8*j;
		}*/
		for (int i = 0; i < 1000000000; i++){
			k=k-1+k^7;
		}
		arr[id] += 1;
	}
}
hipError_t f1(int* arr,int size){
	printf("call f1\n");
	dim3 grid(size / 1024 + 1, 1, 1);
	dim3 block(1024, 1, 1);
	f1_kernel << <grid, block >> >(arr, size);
	//if (hipDeviceSynchronize() != hipSuccess){
	//	printf("cudasync failed\n");
	//	}
	return hipSuccess;
}

__global__ void f2_kernel(float* arr, int size){
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if (id == 1)
		printf("call f2_kernel\n");
	if (id < size){
		arr[id] /=0.111;
	}
	flag = false;
}

hipError_t f2(float* arr, int size){
	//int res[1000];
	//flag = false;
	//hipMemcpy(res, arr, sizeof(int) * 1000, hipMemcpyDeviceToHost);
	//for (int i = 0; i < 1000; i++)
	//{
	//	printf("f%d,", res[i]);
	//}
	printf("call f2\n");
	dim3 grid(size / 1024 + 1, 1, 1);
	dim3 block(1024, 1, 1);
	f2_kernel << <grid, block >> >(arr, size);
	printf("here\n");
	
	/*if (hipDeviceSynchronize() != hipSuccess){
		printf("cudasync failed\n");
	}*/
	return hipSuccess;
}

hipError_t f3(int* arr, int size){
	hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	hipsparseMatDescr_t descr = 0;
	float alpha = 1.0f;
	float beta = 0.0f;
	if (hipsparseCreate(&handle) != HIPSPARSE_STATUS_SUCCESS)
		printf("cusparse create handle failed\n");
	if (hipsparseCreateMatDescr(&descr) != HIPSPARSE_STATUS_SUCCESS)
		printf("cusparse create matrix descr failed\n");

	float* V = new float[size];
	for (int i = 0; i < size; i++){
		V[i] = 2.0f;
	}
	int* csr_col = new int[size];
	for (int i = 0; i < size;i++)
	{
		csr_col[i] = 0;
	}
	int* csr_row = new int[size + 1];
	
	for (int i = 0; i < size + 1;i++)
	{
		csr_row[i] = i;
	}
	float* csr_val = new float[size];
	for (int i = 0; i < size; i++){
		csr_val[i] = 1.0f;
	}
	float* V_dev = 0;
	int* csr_col_dev = 0;
	int* csr_row_dev = 0;
	float* csr_val_dev = 0;
	hipMalloc((void**)&V_dev, sizeof(float)*size);
	hipMalloc((void**)&csr_row_dev, sizeof(int)*(size + 1));
	hipMalloc((void**)&csr_col_dev, sizeof(int)*size);
	hipMalloc((void**)&csr_val_dev, sizeof(float)*size);

	hipMemcpy(V_dev, V, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(csr_row_dev, csr_row, sizeof(int)*(size + 1), hipMemcpyHostToDevice);
	hipMemcpy(csr_col_dev, csr_col, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(csr_val_dev, csr_val, sizeof(float)*size, hipMemcpyHostToDevice);


	f2(V_dev, size);

	if (hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, size, size, size, &alpha, descr, csr_val_dev,
		csr_row_dev, csr_col_dev, V_dev, &beta, V_dev) != HIPSPARSE_STATUS_SUCCESS){
		printf("cusparse csr mv failed\n");
	}

	float* res = new float[size];
	hipMemcpy(res, V_dev, sizeof(float)*size, hipMemcpyDeviceToHost);


	for (int i = 0; i < 100; i++){
		std::cout << res[i] << " ,";
	}
	std::cout << std::endl;


















	return hipSuccess;
}


int test1(){

	f3(NULL, 100000);


	//int len = 10000;
	//int* arr = new int[len];
	//memset(arr, 0, sizeof(int)*len);
	////int arr[1000] = { 0 };
	//int *arr_dev = 0;
	//int *arr1_dev = 0;
	//hipMalloc((void**)&arr_dev, sizeof(int) * len);
	//hipMalloc((void**)&arr1_dev, sizeof(int) * len);
	//hipMemcpy(arr_dev, arr, sizeof(int) * len, hipMemcpyHostToDevice);
	//hipMemcpy(arr1_dev, arr, sizeof(int) * len, hipMemcpyHostToDevice);



	//f1(arr_dev, len);
	//f2(arr1_dev, len);



	//int * res = new int[len];
	////int res[len];
	//hipMemcpy(res, arr1_dev, sizeof(int) * len, hipMemcpyDeviceToHost);
	//for (int i = 0; i < 1000; i++)
	//{
	//	printf("%d,", res[i]);
	//}
	//printf("\n");
	return 0;
}

int main()
{
	

	test1();
	getchar();



	//TestcublasCdotc();

	//TestCuSparse();


	//clock_t start, end;
	//start = clock();
	//hipfftHandle plan;
	//for (int i = 0; i < 10000000000;i++)
	//{
	//	int m=9;
	//	int n;
	//	//n = m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m*m;
	//}
	//getchar();
	//end = clock();
	//printf("%f\n", end - start);




















	hipComplex* input_data = new hipComplex[NX];
	hipComplex* output_data = new hipComplex[NX];
	std::cout << "malloc finished" << std::endl;
	//hipComplex* output_data = new hipComplex[NX];
	for (int i = 0; i < NX; i++)
	{
		input_data[i].x = i;
		input_data[i].y = 0;
		//onput_data[i].y = rand() / 12;
	}

	std::cout << "Initial input_data finished" << std::endl;

	//hipComplex* data_dev = 0;
	//hipMalloc((void**)&data_dev, sizeof(hipComplex)*NX);
	//hipMemcpy(data_dev, input_data, sizeof(hipComplex)*NX, hipMemcpyHostToDevice);
	//memset(input_data, 0, sizeof(hipComplex)*NX);

	//hipMemcpy(input_data, data_dev + 100, sizeof(hipComplex) * 100, hipMemcpyDeviceToHost);

	for (int j = 0; j < 100; j++)
	{
		std::cout << input_data[j].x << std::endl;
	}
	TestFFT(input_data, input_data);
	for (int i = 0; i < 10; i++)
	{
		std::cout << input_data[i].x << "+" << input_data[i].y << std::endl;
	}









	//clock_t start, end;
	//start = clock();
	//TestFFT(input_data, input_data);

	//end = clock();
	//std::cout << "time cost" << end - start << std::endl;

	////std::cout << "Finished/*,Time cost:"/* <<end-start*/<< std::endl;


	//for (int i = 0; i < 10; i++)
	//{
	//	std::cout << input_data[i].x << "+" << input_data[i].y << std::endl;
	//}

	//delete[] input_data;
	//delete[] output_data;
	getchar();
	return 0;
}