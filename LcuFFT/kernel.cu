#include <iostream>
#include "hip/hip_runtime.h"

#include <time.h>
#include "hipfft/hipfft.h"
#define  NX 250000000


//cudaError_t TestFFT(cuComplex* idata, cuComplex* odata);
hipError_t TestFFT(hipComplex* idata,hipComplex* odata)
{
	hipfftHandle plan;
	hipfftComplex *data;
	hipMalloc((void**)&data, sizeof(hipfftComplex)*NX);
	hipMemcpy(data, idata, sizeof(hipfftComplex)*NX,hipMemcpyHostToDevice);
	clock_t start, end;
	start = clock();
	
	if (hipfftPlan1d(&plan,NX,HIPFFT_C2C,1)!=HIPFFT_SUCCESS)
	{
		std::cout << "Plan creation failed" << std::endl;
	}
	
	if (hipfftExecC2C(plan,data,data,HIPFFT_FORWARD)!=HIPFFT_SUCCESS)
	{
		std::cout << "ExecC2C Forward failed" << std::endl;
	}
	hipDeviceSynchronize();
	end = clock();
	std::cout << "time cost" << end - start << std::endl;
	/*if (cudaDeviceSynchronize()!=cudaSuccess)
	{
		std::cout << "Failed to synchronized" << std::endl;
	}*/
	hipMemcpy(odata, data, sizeof(hipfftComplex)*NX, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	return hipSuccess;
}


int main()
{

	hipComplex* input_data = new hipComplex[NX];
	std::cout << "malloc finished" << std::endl;
	//cuComplex* output_data = new cuComplex[NX];
	for (int i = 0; i < NX; i++)
	{
		input_data[i].x = rand() / 10;
		//onput_data[i].y = rand() / 12;
	}

	std::cout << "Initial input_data finished" << std::endl;


	
	TestFFT(input_data, input_data);



	std::cout << "Finished/*,Time cost:"/* <<end-start*/<< std::endl;


	for (int i = 0; i < 10; i++)
	{
		std::cout << input_data[i].x << "+" << input_data[i].y << std::endl;
	}

	delete[] input_data;
	//delete[] output_data;
	getchar();
	return 0;
}